//===----------------------------------------------------------------------===//
//
// Part of CUDA Experimental in CUDA C++ Core Libraries,
// under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
// SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES.
//
//===----------------------------------------------------------------------===//

#include <cuda/experimental/memory_resource>
#include <cuda/std/cstdint>
#include <cuda/std/type_traits>
#include <cuda/stream_ref>

#include <catch2/catch.hpp>

using pool = cuda::experimental::mr::cuda_memory_pool;
static_assert(!cuda::std::is_trivial<pool>::value, "");
static_assert(!cuda::std::is_trivially_default_constructible<pool>::value, "");
static_assert(!cuda::std::is_default_constructible<pool>::value, "");
static_assert(!cuda::std::is_copy_constructible<pool>::value, "");
static_assert(!cuda::std::is_move_constructible<pool>::value, "");
static_assert(!cuda::std::is_copy_assignable<pool>::value, "");
static_assert(!cuda::std::is_move_assignable<pool>::value, "");
static_assert(!cuda::std::is_trivially_destructible<pool>::value, "");
static_assert(!cuda::std::is_empty<pool>::value, "");

static bool ensure_release_threshold(::hipMemPool_t pool, const size_t expected_threshold)
{
  size_t release_threshold = expected_threshold + 1337; // use something different than the expected threshold
  _CCCL_TRY_CUDA_API(
    ::hipMemPoolGetAttribute,
    "Failed to call hipMemPoolGetAttribute",
    pool,
    ::hipMemPoolAttrReleaseThreshold,
    &release_threshold);
  return release_threshold == expected_threshold;
}

static bool ensure_disable_reuse(::hipMemPool_t pool, const int driver_version)
{
  int disable_reuse = 0;
  _CCCL_TRY_CUDA_API(
    ::hipMemPoolGetAttribute,
    "Failed to call hipMemPoolGetAttribute",
    pool,
    ::hipMemPoolReuseAllowOpportunistic,
    &disable_reuse);

  constexpr int min_async_version = 11050;
  return driver_version < min_async_version ? disable_reuse == 0 : disable_reuse != 0;
}

static bool ensure_export_handle(::hipMemPool_t pool, const ::hipMemAllocationHandleType allocation_handle)
{
  size_t handle              = 0;
  const ::hipError_t status = ::hipMemPoolExportToShareableHandle(&handle, pool, allocation_handle, 0);
  ::hipGetLastError(); // Clear CUDA error state

  // If no export was defined we need to querry hipErrorInvalidValue
  return allocation_handle == ::hipMemHandleTypeNone ? status == ::hipErrorInvalidValue : status == ::hipSuccess;
}

TEST_CASE("cuda_memory_pool construction", "[memory_resource]")
{
  int current_device{};
  {
    _CCCL_TRY_CUDA_API(::hipGetDevice, "Failed to querry current device with with hipGetDevice.", &current_device);
  }

  int driver_version = 0;
  {
    _CCCL_TRY_CUDA_API(::hipDriverGetVersion, "Failed to call hipDriverGetVersion", &driver_version);
  }

  ::hipMemPool_t current_default_pool{};
  {
    _CCCL_TRY_CUDA_API(::hipDeviceGetDefaultMemPool,
                       "Failed to call hipDeviceGetDefaultMemPool",
                       &current_default_pool,
                       current_device);
  }

  using memory_pool = cuda::experimental::mr::cuda_memory_pool;
  SECTION("Construct from device id")
  {
    cuda::experimental::mr::cuda_memory_pool from_device{current_device};

    ::hipMemPool_t pool_handle = from_device.pool_handle();
    CHECK(pool_handle != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(pool_handle, 0));

    // Ensure that we disable reuse with unsupported drivers
    CHECK(ensure_disable_reuse(pool_handle, driver_version));

    // Ensure that we disable export
    CHECK(ensure_export_handle(pool_handle, ::hipMemHandleTypeNone));
  }

  SECTION("Construct with empty properties")
  {
    cuda::experimental::mr::cuda_memory_pool_properties props{};
    memory_pool from_defaulted_properties{current_device, props};

    ::hipMemPool_t pool_handle = from_defaulted_properties.pool_handle();
    CHECK(pool_handle != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(pool_handle, 0));

    // Ensure that we disable reuse with unsupported drivers
    CHECK(ensure_disable_reuse(pool_handle, driver_version));

    // Ensure that we disable export
    CHECK(ensure_export_handle(pool_handle, ::hipMemHandleTypeNone));
  }

  SECTION("Construct with initial pool size")
  {
    cuda::experimental::mr::cuda_memory_pool_properties props = {42, 20};
    memory_pool with_threshold{current_device, props};

    ::hipMemPool_t pool_handle = with_threshold.pool_handle();
    CHECK(pool_handle != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(pool_handle, props.release_threshold));

    // Ensure that we disable reuse with unsupported drivers
    CHECK(ensure_disable_reuse(pool_handle, driver_version));

    // Ensure that we disable export
    CHECK(ensure_export_handle(pool_handle, ::hipMemHandleTypeNone));
  }

  // Allocation handles are only supported after 11.2
#if !defined(_CCCL_CUDACC_BELOW_11_2)
  SECTION("Construct with allocation handle")
  {
    cuda::experimental::mr::cuda_memory_pool_properties props = {
      42, 20, cuda::experimental::mr::hipMemAllocationHandleType::hipMemHandleTypePosixFileDescriptor};
    memory_pool with_allocation_handle{current_device, props};

    ::hipMemPool_t pool_handle = with_allocation_handle.pool_handle();
    CHECK(pool_handle != current_default_pool);

    // Ensure we use the right release threshold
    CHECK(ensure_release_threshold(pool_handle, props.release_threshold));

    // Ensure that we disable reuse with unsupported drivers
    CHECK(ensure_disable_reuse(pool_handle, driver_version));

    // Ensure that we disable export
    CHECK(ensure_export_handle(pool_handle, static_cast<hipMemAllocationHandleType>(props.allocation_handle_type)));
  }
#endif // !_CCCL_CUDACC_BELOW_11_2
}

TEST_CASE("cuda_memory_pool comparison", "[memory_resource]")
{
  int current_device{};
  {
    _CCCL_TRY_CUDA_API(::hipGetDevice, "Failed to querry current device with with hipGetDevice.", &current_device);
  }

  int driver_version = 0;
  {
    _CCCL_TRY_CUDA_API(::hipDriverGetVersion, "Failed to call hipDriverGetVersion", &driver_version);
  }

  ::hipMemPool_t current_default_pool{};
  {
    _CCCL_TRY_CUDA_API(::hipDeviceGetDefaultMemPool,
                       "Failed to call hipDeviceGetDefaultMemPool",
                       &current_default_pool,
                       current_device);
  }

  cuda::experimental::mr::cuda_memory_pool first{current_device};
  { // comparison against a plain cuda_memory_pool
    cuda::experimental::mr::cuda_memory_pool second{current_device};
    CHECK(first == first);
    CHECK(first != second);
  }

  { // comparison against a hipMemPool_t
    CHECK(first == first.pool_handle());
    CHECK(first.pool_handle() == first);
    CHECK(first != current_default_pool);
    CHECK(current_default_pool != first);
  }
}
